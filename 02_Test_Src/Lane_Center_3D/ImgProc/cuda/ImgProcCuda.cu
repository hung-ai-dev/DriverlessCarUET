#include "hip/hip_runtime.h"
#include "ImgProcCuda.h"

int divUp(int a, int b){ return (a + b - 1) / b; }

__device__ float dot(const float3& v1, const float3& v2)
{
	return __fmaf_rn(v1.x, v2.x, __fmaf_rn(v1.y, v2.y, v1.z*v2.z));
}

__device__ float3 cross(const float3& v1, const float3& v2)
{
	return make_float3(v1.y * v2.z - v1.z * v2.y, v1.z * v2.x - v1.x * v2.z, v1.x * v2.y - v1.y * v2.x);
}

__device__ float3 operator+(const float3& v1, const float3& v2)
{
	return make_float3(v1.x + v2.x, v1.y + v2.y, v1.z + v2.z);
}

__device__ float3 operator-(const float3& v1, const float3& v2)
{
	return make_float3(v1.x - v2.x, v1.y - v2.y, v1.z - v2.z);
}

__global__ void kernel_convert_Depth_To_Point3f(cv::cuda::PtrStep<unsigned short> _depth, const ImgProc3D::Intr cam, cv::cuda::PtrStep<float3> _point3f)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float d = float(_depth(y, x)) / cam.scale;
	_point3f(y, x) = { (x - cam.cx) * d / cam.fx, (y - cam.cy) * d / cam.fy, d };
	return;
}

__global__ void kernel_GenGridMap2D(cv::cuda::PtrStep<float3> _point3f, cv::cuda::PtrStep<uchar3> _rgb,
	float4 pModel,
	float3 pOrg, float3 e_1, float3 e_2,
	cv::cuda::PtrStep<uchar3> _map)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float3 p = _point3f(y, x);

	float3 p_new = p - pOrg;
	float p_x_new = dot(e_1, p_new);
	float p_y_new = dot(e_2, p_new);

	int new_x = int(LANE_MAP_SIZE / 2 + p_x_new * LANE_MAP_SCALE);
	int new_y = int(LANE_MAP_SIZE - p_y_new * LANE_MAP_SCALE);

	if (fabs(pModel.x*p.x + pModel.y*p.y + pModel.z*p.z + pModel.w) < 0.05f)
	{
		if (new_x > 0 && new_x < LANE_MAP_SIZE  &&  new_y > 0 && new_y < LANE_MAP_SIZE)
		{
			_map(new_y, new_x) = _rgb(y, x);
		}
	}
	else
	{
		if (new_x > 0 && new_x < LANE_MAP_SIZE  &&  new_y > 0 && new_y < LANE_MAP_SIZE)
		{
			_map(new_y, new_x) = { 0, 0, 255 };
		}
	}

	return;
}

void caller_convertTo_Point3fMap(cv::cuda::PtrStep<unsigned short> _depth, const ImgProc3D::Intr cam, cv::cuda::PtrStep<float3> _point3f, int width, int height)
{
	/*cv::Size sz = depth.size();*/
	dim3 block(32, 16);
	dim3 grid(divUp(width, block.x), divUp(height, block.y));
	kernel_convert_Depth_To_Point3f << <grid, block >> >(_depth, cam, _point3f);
}

void caller_GenGridMap2D(cv::cuda::PtrStep<float3> _point3f, cv::cuda::PtrStep<uchar3> _rgb,
	float4 pModel,
	float3 pOrg, float3 e_1, float3 e_2,
	cv::cuda::PtrStep<uchar3> _map, int width, int height)
{
	dim3 block(32, 16);
	dim3 grid(divUp(width, block.x), divUp(height, block.y));
	kernel_GenGridMap2D << <grid, block >> >(_point3f, _rgb, pModel, pOrg, e_1, e_2, _map);
}